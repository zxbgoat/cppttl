#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "common.h"


__global__ void vectoradd(const float *A, const float *B, const float *C, int num)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num) c[i] = A[i] + B[i];
}


int main()
{
    hipError_t err = hipSuccess;
    int num = 50000;
    size_t size = num * sizeof(float);
    printf("[Vector addition of %d elements]\n", num);
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < num; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    float *d_A = NULL;
    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float *d_B = NULL;
    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float *d_C = NULL;
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int threadsPerBlock = 256;
    int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectoradd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, num);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < num; ++i)
    {
        if (fabs(h_A[i]+h_B[i]) - h_C[i] > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");
    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    printf("Done\n");
    return 0;
}
